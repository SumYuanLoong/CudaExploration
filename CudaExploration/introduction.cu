﻿/* 
This follows the guide provided on https://developer.nvidia.com/blog/even-easier-introduction-cuda/
Any clarifications needed pls refer to the guide
*/

#include <iostream>
#include <math.h>
#include "hip/hip_runtime.h"               //headers for the cuda methods


// function to add the elements of two arrays
__global__ void add(int n, float* x, float* y) 
/*"__global__"" to declare this as a method to be executed in cuda
    this is known as a kernal
    GPU code is called device code and CPU code is called host code
*/
{
    for (int i = 0; i < n; i++)
        y[i] = x[i] + y[i];
}

int main(void)
{
    int N = 1 << 20; // 1M elements, << operation is bitwise shift

    //float* x = new float[N];
    //float* y = new float[N];
    float* x, * y;
    hipMallocManaged(&x, N * sizeof(float));       //allocation of unified memory, addresses that are accessible from both gpu and cpu
    hipMallocManaged(&y, N * sizeof(float));       // note that the actual location of the data is managed by nvcc which automatically handles the copying of data to and from the gpu

    // initialize x and y arrays on the host
    for (int i = 0; i < N; i++) {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    // Run kernel on 1M elements on the GPU
    add <<<1, 1 >>> (N, x, y);

    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();

    // Check for errors (all values should be 3.0f)
    float maxError = 0.0f;
    for (int i = 0; i < N; i++)
        maxError = fmax(maxError, fabs(y[i] - 3.0f));
    std::cout << "Max error: " << maxError << std::endl;

    // Free memory
    hipFree(x);
    hipFree(y);

    return 0;
}