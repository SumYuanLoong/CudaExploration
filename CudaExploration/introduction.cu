﻿/* 
This follows the guide provided on https://developer.nvidia.com/blog/even-easier-introduction-cuda/
Any clarifications needed pls refer to the guide
*/

#include <iostream>
#include <math.h>
#include <time.h>
#include "hip/hip_runtime.h"               //headers for the cuda methods


// function to add the elements of two arrays
__global__ void add(int n, float* x, float* y) 
/*"__global__"" to declare this as a method to be executed in cuda
    this is known as a kernal
    GPU code is called device code and CPU code is called host code
*/
{
    //int index = threadIdx.x; //threadIdx returns the index of the current thread
    //int stride = blockDim.x; //blockDim returns the number of threads in the current block

    int index = blockIdx.x * blockDim.x + threadIdx.x;  //block index * threads per block * thread index
    int increment = blockDim.x * gridDim.x;             //threads per block * total threads active
    for (int i = index; i < n; i += increment) {
		y[i] = x[i] + y[i];
		//printf("%d\n", i);
    }
}

int main(void)
{
    printf("Process started");
    
    int N = 1 << 20; // 1M elements, << operation is bitwise shift
    //float* x = new float[N];
    //float* y = new float[N];
    float* x, * y;
    hipMallocManaged(&x, N * sizeof(float));       //allocation of unified memory, addresses that are accessible from both gpu and cpu
    hipMallocManaged(&y, N * sizeof(float));       // note that the actual location of the data is managed by nvcc which automatically handles the copying of data to and from the gpu

    // initialize x and y arrays on the host
    for (int i = 0; i < N; i++) {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    int blockSize = 256;
    int numBlocks = (N + blockSize - 1) / blockSize; // calculation of the amount of blocks of 256 needed to complete the task
    clock_t tstart = clock();
    // Run kernel on 1M elements on the GPU
    add <<<numBlocks, 256 >>> (N, x, y);
    /* The key code here is the <<< blocks , threads >>> that tells the compiler this code is meant to run on the GPU
    Threads have to be in a multiple of 32, maximum of 1024*/

    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();

    printf("Time taken: %.9fs\n", (double)(clock() - tstart) / CLOCKS_PER_SEC);

    // Check for errors (all values should be 3.0f)
    float maxError = 0.0f;
    for (int i = 0; i < N; i++)
        maxError = fmax(maxError, fabs(y[i] - 3.0f));
    std::cout << "Max error: " << maxError << std::endl;

    // Free memory
    hipFree(x);
    hipFree(y);

    return 0;
}