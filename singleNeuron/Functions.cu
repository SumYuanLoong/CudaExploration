
#include <hip/hip_runtime.h>
﻿__global__ void linearRegress(int maxrows, float* pdataset, double* pzArr, int col) {
    //maxRows = sizeof(TrainSetData) / sizeof(*pTrainSetData);

    int index = blockIdx.x * blockDim.x + threadIdx.x;  //block index * threads per block * thread index
    //int increment = blockDim.x * gridDim.x;             //threads per block * total threads active

    int column = index % 9;     //determine what weight to apply 
    int row = index / 9;        //to determine where to save z to



    //int a, b, c = 0; // a is loop counter, b for position in row, c for row
    //double z = 0;
    //for (a = 0, b = 0; a < maxrows; a++, pdataset++) {
    //    if (b == (col - 2)) {
    //        z += (weight[b] * *pdataset) + bias;
    //        *pzArr = z;
    //        pzArr++;   // increment to next value in z arrary
    //        b = 0;
    //        z = 0;        //reset column and z row values
    //    }
    //    else
    //    {
    //        z += (weight[b++] * *pdataset);     //calculation
    //    }
    //}
}
//
////sigmoid function taking in the z arr
//void sigmoid(double zArr[], double sigArr[], int arrSz) {
//    int i;
//    for (i = 0; i < arrSz; i++) {
//        sigArr[i] = (1 / (1 + exp(-zArr[i])));
//    }
//}
//
////computing the mmse function for the testing and training sets
//void mmseFunc(double* trainMmse, double* testMmse) {
//    int i = 0;
//    double mmsesum = 0;
//
//    for (i = 0; i < trRow; i++) {
//        mmsesum += (pow((trainsig[i] - TrainSetDiag[i]), 2));
//    }
//    *trainMmse = mmsesum / trRow;
//    mmsesum = 0;
//    for (i = 0; i < tsRow; i++) {
//        mmsesum += (pow((testsig[i] - TestSetDiag[i]), 2));
//    }
//    *testMmse = mmsesum / tsRow;
//}

//calculation of mae for training set mae is only dependent on training set
double maeFunc(int trRow, float *sigma, float *diag) {
    int i;
    double maesum = 0;
    for (i = 0; i < trRow; i++) {
        maesum += fabs(*sigma - *diag);
    }
    return maesum / 90;
}

////backPropagate
//void backPropagate() {
//    int x, y;
//    double sumtrainw = 0, sumtrainb = 0;
//    for (y = 0; y < col - 1; y++)
//    {
//        //printf("\nsummation z[%d] = %f", a, trainz[a]);
//        //printf("\nsigmoid y[%d] = %f d[%d] = %f", a, trainsig[a], a, trainoutpdata[a][0]);
//        //printf("\nuntrained mmse (1*(summation ycap - d)^2)/90 = %f", ummse);
//        //printf("\nmae (1*(summation ycap - d))/90 = %f\n", mae);
//
//        for (x = 0; x < trRow; x++)
//        {
//            sumtrainw += (trainsig[x] - TrainSetDiag[x]) * (exp(trainz[x]) / ((1 + exp(trainz[x])) * (1 + exp(trainz[x]))) * TrainSetData[x][y]);
//            //printf("\nsumtrainw[%d][%d] = %f", a, b, sumtrainw);
//            if (y == 8)
//            {
//                sumtrainb += (trainsig[x] - TrainSetDiag[x]) * (exp(trainz[x]) / ((1 + exp(trainz[x])) * (1 + exp(trainz[x]))) * 1);
//                //printf("\nsumtrainb[%d][%d] = %f", x, y, sumtrainb);
//
//            }
//        }
//        sumtrainw = (sumtrainw / trRow);
//        weight[y] = (weight[y] - (trainspeed * sumtrainw)); //update the new weight into oldw[0-8]
//        //printf("\ntrainedw[%d][%d] = %f", x, y, weight[y]);
//        sumtrainb = (sumtrainb / trRow);
//        bias = (bias - (trainspeed * sumtrainb)); //update the new bias b into oldb
//        sumtrainw = 0;
//        sumtrainb = 0;
//    }
//}