#include "hip/hip_runtime.h"
﻿#include <iostream>
#include <math.h>
#include <time.h>
#include <conio.h>
#include "hip/hip_runtime.h"               //headers for the cuda methods
#include ""
#include "Functions.cuh"

#define TMAE 0.15
#define trainspeed 0.05
#define totalRows 100
#define trRow 90 //number of rows in the training set
#define tsRow 10 //number of rows in the test set
#define col 10 //columns of data including desired value "result"

int trainingDataCount = trRow * (col - 1);
int testDataCount = tsRow * (col - 1);

//datasets
float TrainSetData[trRow][col - 1]; 
float TestSetData[tsRow][col - 1]; 
float TrainSetDiag[trRow]; //training result set 
float TestSetDiag[tsRow]; //testing result set
double trainz[trRow]; //store training set z value of each patient
double testz[tsRow]; //store testing set z value of each patient
double trainsig[trRow]; //store training set sigmoid y cap of each patient
double testsig[tsRow]; //store testing set sigmoid y cap of each patient

//pointers to the datasets

//training data set
//data is r1c1 r1c2 r1c9 r2c1 r2c2
//1 row is 1 patient
float* pTrainSetData;
//testing data set
//data is r1c1 r1c2 r1c9 r2c1 r2c2
//1 row is 1 patient
float* pTestSetData;
float* pTrainSetDiag;
float* pTestSetDiag;
double* ptrainz;
double* ptestz;

//original data/weights/bias for printing at end
double weight[9];
double bias;
double utrmmse, utsmmse, ttrmmse, ttsmmse;
double* putrmmse = &utrmmse;
double* putsmmse = &utsmmse;
double* pttrmmse = &ttrmmse;
double* pttsmmse = &ttsmmse;

void readFile(float* traindata, float* testdata, float* trainDiag, float* testDiag);
double random();
void matrix();

int main(void) {
    clock_t tstart = clock(); //start clock
    srand(time(NULL));
    
    //cuda memory allocation
    hipMallocManaged(&pTrainSetData, trRow * (col - 1) * sizeof(float));
    hipMallocManaged(&pTestSetData, tsRow * (col - 1) * sizeof(float));
    hipMallocManaged(&pTrainSetDiag, trRow  * sizeof(float));
    hipMallocManaged(&pTestSetDiag, tsRow  * sizeof(float));
    hipMallocManaged(&ptrainz, trRow * sizeof(double));
    hipMallocManaged(&ptestz, trRow * sizeof(double));

    readFile(pTrainSetData, pTestSetData, pTrainSetDiag, pTestSetDiag);

    
    int numBlocks = (trainingDataCount + 256 - 1) / 256;
    memset(ptrainz, 0, trRow * sizeof(double));  // set the z arr to 0 so the threads can assign values
    linearRegress <<<numBlocks, 256 >>> (trRow, pTrainSetData, ptrainz, col);
    
}

void readFile(float *traindata, float *testdata, float *trainDiag, float *testDiag) {
    int x, y;
    int a=0, b=0, c=0, d=0;
    FILE* fertfile_ptr = fopen("fertility_Diagnosis_Data_Group1_4.txt", "r");

    // error handling
    if (fertfile_ptr == NULL)
    {
        fprintf(stderr, "Error opening file: ");
        exit(EXIT_FAILURE);
    }

    for (x = 0; x < totalRows; x++) {
        for (y = 0; y < col; y++) {
            if (y == (col - 1)) { //result of diagnosis
                if (x < trRow) {
                    fscanf(fertfile_ptr, "%f, ", trainDiag);
                    trainDiag++,a++;
                }
                else {
                    fscanf(fertfile_ptr, "%f, ", testDiag);
                    testDiag++,b++;
                }
            }
            else {  //data to determine diagnosis
                if (x < trRow) {
                    fscanf(fertfile_ptr, "%f, ", traindata);
                    traindata++,c++;
                }
                else {
                    fscanf(fertfile_ptr, "%f, ", testdata);
                    testdata++,d++;
                }
            }
        }
    }
    fclose(fertfile_ptr);
    printf("%d training data read.\n", c);
    printf("%d training diag read.\n", a);

    printf("%d testing data read.\n", d);
    printf("%d testing diag read.\n", b);


}

//generate a number between -1 and 1
double random()
{
    int w;
    double resultrand;
    w = (rand() % 3) - 1; //random between int -1, 0 , 1
    if (w > 1 || w < -1)
    {
        w = (rand() % 3) - 1; //random between int -1, 0 , 1
        //printf("%d", w);
    }
    if (w == 0)
        w = 1;
    //to improve the random result for double -1.00 to 1.00 by using w
    resultrand = (1.0 * rand() / RAND_MAX - w);
    if (resultrand > 1.00)
    {
        resultrand = resultrand - 1;
    }
    //printf("\nweight = %lf", resultrand);
    return resultrand;
}

// to display the confusion matrix
void matrix() {
    int tp = 0, fp = 0, tn = 0, fn = 0, i, y;
    for (i = 0; i < trRow; i++) {
        y = round(trainsig[i]);
        if (y == 1)
        {
            if (TrainSetDiag[i] == y)
                tp++;
            else
                fp++;
        }
        else
        {
            if (TrainSetDiag[i] == y)
                tn++;
            else
                fn++;
        }
    }
    printf("\n-------------------------------------------\n\n");
    printf("Training Set Confusion Matrix\n                          True      False\n");
    printf("Predicted Positive        %d         %d\n", tp, fp);
    printf("Predicted Negative        %d        %d\n", tn, fn);
    printf("\n-------------------------------------------\n\n");
    tp = 0, fp = 0, tn = 0, fn = 0;

    for (i = 0; i < tsRow; i++) {
        y = round(testsig[i]);
        if (y == 1)
        {
            if (TestSetDiag[i] == y)
                tp++;
            else
                fp++;
        }
        else
        {
            if (TestSetDiag[i] == y)
                tn++;
            else
                fn++;
        }
    }
    printf("Testing Set Confusion Matrix\n                          True      False\n");
    printf("Predicted Positive        %d         %d\n", tp, fp);
    printf("Predicted Negative        %d         %d", tn, fn);
    printf("\n\n-------------------------------------------\n\n");
}