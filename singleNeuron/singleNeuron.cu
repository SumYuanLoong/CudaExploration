#include "hip/hip_runtime.h"
﻿#include <iostream>
#include <math.h>
#include <time.h>
#include <conio.h>
#include "hip/hip_runtime.h"               //headers for the cuda methods
#include ""
#include "Functions.cuh"

#define TMAE 0.15
#define trainspeed 0.05
#define totalRows 100
#define trRow 90 //number of rows in the training set
#define tsRow 10 //number of rows in the test set
#define col 10 //columns of data including desired value "result"

//datasets
float TrainSetData[trRow][col - 1]; //training data set
float TestSetData[tsRow][col - 1]; //testing data set
float TrainSetDiag[trRow]; //training result set 
float TestSetDiag[tsRow]; //testing result set
double trainz[trRow]; //store training set z value of each patient
double testz[tsRow]; //store testing set z value of each patient
double trainsig[trRow]; //store training set sigmoid y cap of each patient
double testsig[tsRow]; //store testing set sigmoid y cap of each patient

//pointers to the datasets
float* pTrainSetData = &TrainSetData[0][0];
float* pTestSetData = &TestSetData[0][0];
float* pTrainSetDiag = &TrainSetDiag[0];
float* pTestSetDiag = &TestSetDiag[0];
double* ptrainz = &trainz[0];
double* ptestz = &testz[0];

//original data/weights/bias for printing at end
double weight[9];
double bias;
double utrmmse, utsmmse, ttrmmse, ttsmmse;
double* putrmmse = &utrmmse;
double* putsmmse = &utsmmse;
double* pttrmmse = &ttrmmse;
double* pttsmmse = &ttsmmse;

void readFile();

int main(void) {
    clock_t tstart = clock(); //start clock
    srand(time(NULL));
    //everything in between
    readFile();

    
}

void readFile() {
    int x, y;
    FILE* fertfile_ptr = fopen("fertility_Diagnosis_Data_Group1_4.txt", "r");

    // error handling
    if (fertfile_ptr == NULL)
    {
        fprintf(stderr, "Error opening file: ");
        exit(EXIT_FAILURE);
    }

    for (x = 0; x < totalRows; x++) {
        for (y = 0; y < col; y++) {
            if (y == (col - 1)) { //result of diagnosis
                if (x < trRow) {
                    fscanf(fertfile_ptr, "%f, ", &TrainSetDiag[x]);
                }
                else {
                    fscanf(fertfile_ptr, "%f, ", &TestSetDiag[x - trRow]);
                }
            }
            else {  //data to determine diagnosis
                if (x < trRow) {
                    fscanf(fertfile_ptr, "%f, ", &TrainSetData[x][y]);
                }
                else {
                    fscanf(fertfile_ptr, "%f, ", &TestSetData[x - trRow][y]);
                }
            }
        }
    }
    fclose(fertfile_ptr);
}